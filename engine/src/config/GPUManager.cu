#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"

long gpuMemorySize() {
	int gpuId = 0;
	
	// To get the total size of the current
	struct hipDeviceProp_t props;
	CheckCudaErrors( hipSetDevice(gpuId) );
	hipGetDeviceProperties(&props, gpuId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return total;
}
