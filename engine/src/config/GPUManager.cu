#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"


namespace ral {
namespace config {

size_t gpuMemorySize() {
	// NOTE if CUDA_VISIBLE_DEVICES is 6 and we use 0 here it means we take 6
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	CheckCudaErrors( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

}	// namespace config
}	// namespace ral
