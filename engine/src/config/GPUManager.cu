#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudf/utilities/error.hpp>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"

namespace ral {
namespace config {

size_t gpuMemorySize() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}
size_t gpuUsedMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return total - free;
}

}	// namespace config
}	// namespace ral
