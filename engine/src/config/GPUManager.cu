#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"


namespace ral {
namespace config {

GPUManager::GPUManager() : currentDeviceId{0} {
}

int GPUManager::getDeviceId() {
	return this->currentDeviceId;
}


GPUManager& GPUManager::getInstance() {
	static GPUManager instance;
	return instance;
}

void GPUManager::initialize(int deviceId) {
	currentDeviceId = deviceId;
	setDevice();
}

void GPUManager::setDevice() {
	//CheckCudaErrors( hipSetDevice(currentDeviceId) );
}

size_t GPUManager::gpuMemorySize() {
	struct hipDeviceProp_t props;
	CheckCudaErrors( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

}	// namespace config
}	// namespace ral
